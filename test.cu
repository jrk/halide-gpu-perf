#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <assert.h>

#include "test_buffer.h"

const int kernel_radius = 9;
const int kernel_area = (kernel_radius*2+1)*(kernel_radius*2+1);
const int width = 6400,
          height = 4800;

#ifdef _MSC_VER
#define ALWAYS_INLINE __forceinline
#else
#define ALWAYS_INLINE __attribute__((always_inline))
#endif

template <typename T, bool static_addr=false>
ALWAYS_INLINE inline
__device__ T& write_pixel(buffer_t buf, int x, int y) {
    T *data = (T*)buf.dev;
    int x_min, y_min, x_stride, y_stride;
    if (static_addr) {
        assert(buf.stride[0] == 1);
        assert(buf.stride[1] > 6300 && buf.stride[1] < 6500);
        x_min = buf.min[0];
        y_min = buf.min[1];
        x_stride = 1;
        y_stride = buf.stride[1];
    } else {
        x_min = 0;
        y_min = 0;
        x_stride = 1;
        y_stride = width;
    }
    int x_offset = (x - x_min) * x_stride;
    int y_offset = (y - y_min) * y_stride;
    return data[x_offset + y_offset];
}

template <typename T, bool static_addr=false>
ALWAYS_INLINE inline
const __device__ T read_pixel(const buffer_t buf, int x, int y) {
    const T *data = (const T*)buf.dev;
    int x_min, y_min, x_stride, y_stride;
    if (static_addr) {
        assert(buf.stride[0] == 1);
        assert(buf.stride[1] > 6300 && buf.stride[1] < 6500);
        x_min = buf.min[0];
        y_min = buf.min[1];
        x_stride = 1;
        y_stride = buf.stride[1];
    } else {
        x_min = -kernel_radius;
        y_min = -kernel_radius;
        x_stride = 1;
        y_stride = width+2*kernel_radius;
    }
    int x_offset = (x - x_min) * x_stride;
    int y_offset = (y - y_min) * y_stride;
    return __ldg(data + x_offset + y_offset);
}

__global__ void
boxBlurBuf(const buffer_t in, buffer_t out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x + out.min[0];
    int y = blockIdx.y * blockDim.y + threadIdx.y + out.min[1];

#if 0
    if (x < out.min[0] || y < out.min[1] ||
        x >= out.extent[0] || y >= out.extent[1])
    {
        return;
    }
#endif
    
    float res = 0;
    for (int j = -kernel_radius; j <= kernel_radius; j++) {
        for (int i = -kernel_radius; i <= kernel_radius; i++) {
            res += read_pixel<float>(in, x+i, y+j);
        }
    }
    res /= float(kernel_area);

    write_pixel<float>(out, x,y) = res;
}

__global__ void
boxBlurBufStatic(const buffer_t in, buffer_t out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x + out.min[0];
    int y = blockIdx.y * blockDim.y + threadIdx.y + out.min[1];

    float res = 0;
    for (int j = -kernel_radius; j <= kernel_radius; j++) {
        for (int i = -kernel_radius; i <= kernel_radius; i++) {
            res += read_pixel<float,true>(in, x+i, y+j);
        }
    }
    res /= kernel_area;

    write_pixel<float,true>(out, x,y) = res;
}

// TODO: restrict on in/out DOUBLES performance!
#define OUT_PIXEL(x,y) (out[(x)+width*(y)])
#define IN_PIXEL(x,y) (in[((x)+kernel_radius)+width*((y)+kernel_radius)])
__global__ void
boxBlurStatic(const float * __restrict__ in, float *out) {
// boxBlurStatic(const float *in, float *out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float res = 0;
    for (int j = -kernel_radius; j <= kernel_radius; j++) {
        for (int i = -kernel_radius; i <= kernel_radius; i++) {
            res += IN_PIXEL(x+i, y+j);
        }
    }
    res /= float(kernel_area);

    OUT_PIXEL(x,y) = res;
}

__global__ void
boxBlurStaticNonRestrict(const float *in, float *out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float res = 0;
    for (int j = -kernel_radius; j <= kernel_radius; j++) {
        for (int i = -kernel_radius; i <= kernel_radius; i++) {
            res += IN_PIXEL(x+i, y+j);
        }
    }
    res /= float(kernel_area);

    OUT_PIXEL(x,y) = res;
}
#undef OUT_PIXEL
#undef IN_PIXEL

#ifndef __CUDA_ARCH__

const int block_width = 32,
          block_height = 32;
dim3 blocks((width + block_width - 1) / block_width,
            (height + block_height - 1) / block_height);
dim3 threads(block_width, block_height);

using std::vector;
using std::string;
using std::pair;

// template<class ...Args>
// void variant(std::string name, void(*kernel)(Args...), Args... args) {
//     variants.push_back(
//         std::make_pair(name, [&]{
//             kernel<<<blocks, threads>>>(args...);
//         })
//     );
// }
#define variant(nm,...) (variants.push_back(std::make_pair( \
#nm, [&]{\
    (nm)<<<blocks,threads>>>(__VA_ARGS__); \
} \
)))

int main (int argc, char const *argv[])
{
    int trials = 1;
    if (argc == 2) {
        trials = atoi(argv[1]);
    }
    Buffer<float> in(width+2*kernel_radius, height+2*kernel_radius),
               out(width, height);

    in.set_min(-kernel_radius, -kernel_radius);

    in.for_each_element([&](int x, int y) {
        in(x, y) = (x % 3 == 0 && y % 3 == 0) ? 1.f : 0.f;
    });
    
    dev_malloc(in);
    dev_malloc(out);
    
    host_to_dev(in);
    host_to_dev(out);

    hipEvent_t startEv, endEv;
    hipEventCreate(&startEv);
    hipEventCreate(&endEv);
    
    typedef std::function<void(void)> Fn;
    vector<pair<string,Fn> > variants;
    
    variant(boxBlurBuf,
            *(in.raw_buffer()), *(out.raw_buffer()) );
    
    variant(boxBlurBufStatic,
            *(in.raw_buffer()), *(out.raw_buffer()) );
    
    variant(boxBlurStatic,
            (float*)in.raw_buffer()->dev, (float*)out.raw_buffer()->dev);
    
    variant(boxBlurStaticNonRestrict,
            (float*)in.raw_buffer()->dev, (float*)out.raw_buffer()->dev);

    for (auto &variant : variants)
    {
        std::string name;
        Fn fn;
        std::tie(name, fn) = variant;
        
        hipEventRecord(startEv);
        for (int i = 0; i < trials; i++) {
            fn();
        }
        hipEventRecord(endEv);

        dev_to_host(in);
        dev_to_host(out);

        float elapsed;
        hipEventElapsedTime(&elapsed, startEv, endEv);
        printf( "\n-------\n"
                "%s\n"
                "TIME: %f ms / %d trials = %f ms\n",
                name.c_str(), elapsed, trials, elapsed/trials );
        int64_t pixels = width*height;
        int64_t kernel_pixels = (kernel_radius*2+1)*(kernel_radius*2+1);
        printf("Inputs accumulated: %ldM\n", pixels*kernel_pixels/1000000);
    }

    dev_free(in);
    dev_free(out);
    
    return 0;
}
#endif //host-only